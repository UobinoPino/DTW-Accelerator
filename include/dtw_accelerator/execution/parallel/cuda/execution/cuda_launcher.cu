#include "cuda_launcher.hpp"
#include "../core/cuda_memory.hpp"
#include "../kernels/matrix_kernels.cuh"
#include "../kernels/dtw_core_kernels.cuh"
#include "../kernels/path_kernels.cuh"
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

namespace dtw_accelerator {
    namespace parallel {
        namespace cuda {

            using memory::DeviceBuffer;

            template<distance::MetricType M>
            DTWResult dtw_cuda_impl_template(
                    const DoubleTimeSeries& A,
                    const DoubleTimeSeries& B,
                    int tile_size) {

                int n = A.size();
                int m = B.size();
                int dim = A.dimensions();

                DeviceBuffer<double> d_A(n * dim);
                DeviceBuffer<double> d_B(m * dim);
                DeviceBuffer<double> d_D((n + 1) * (m + 1));

                int max_path_length = n + m;
                DeviceBuffer<int> d_path_i(max_path_length);
                DeviceBuffer<int> d_path_j(max_path_length);
                DeviceBuffer<int> d_path_length(1);

                hipMemcpyAsync(d_A.data, A.data(), n * dim * sizeof(double), hipMemcpyHostToDevice);
                hipMemcpyAsync(d_B.data, B.data(), m * dim * sizeof(double), hipMemcpyHostToDevice);

                constexpr int BLOCK_SIZE = 256;
                int grid_size = ((n+1)*(m+1) + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernels::init_cost_matrix<<<grid_size, BLOCK_SIZE>>>(d_D.data, n, m);

                int n_tiles = (n + tile_size - 1) / tile_size;
                int m_tiles = (m + tile_size - 1) / tile_size;
                int total_waves = n_tiles + m_tiles - 1;

                size_t shared_mem_size = 2 * (tile_size + 1) * sizeof(double);

                for (int wave = 0; wave < total_waves; ++wave) {
                    int tiles_in_wave = 0;

                    for (int ti = 0; ti < n_tiles; ++ti) {
                        int tj = wave - ti;
                        if (tj >= 0 && tj < m_tiles) {
                            tiles_in_wave++;
                        }
                    }

                    if (tiles_in_wave > 0) {
                        constexpr int THREADS_PER_BLOCK = 256;
                        dim3 grid(tiles_in_wave, 1, 1);
                        dim3 block(THREADS_PER_BLOCK, 1, 1);

                        if (tile_size == 32) {
                            kernels::dtw_tile_wavefront<M, 32>
                            <<<grid, block, shared_mem_size>>>(
                                    d_D.data, d_A.data, d_B.data, n, m, dim,
                                            wave, n_tiles, m_tiles);
                        } else if (tile_size == 64) {
                            kernels::dtw_tile_wavefront<M, 64>
                            <<<grid, block, shared_mem_size>>>(
                                    d_D.data, d_A.data, d_B.data, n, m, dim,
                                            wave, n_tiles, m_tiles);
                        } else {
                            kernels::dtw_tile_wavefront<M, 32>
                            <<<grid, block, shared_mem_size>>>(
                                    d_D.data, d_A.data, d_B.data, n, m, dim,
                                            wave, n_tiles, m_tiles);
                        }
                    }
                }

                kernels::backtrack_path<<<1, 1>>>(
                        d_D.data, d_path_i.data, d_path_j.data,
                                d_path_length.data, n, m);

                int h_path_length;
                hipMemcpyAsync(&h_path_length, d_path_length.data, sizeof(int),
                                hipMemcpyDeviceToHost);

                if (h_path_length > 1) {
                    int reverse_blocks = (h_path_length / 2 + 255) / 256;
                    kernels::reverse_path<<<reverse_blocks, 256>>>(
                            d_path_i.data, d_path_j.data, h_path_length);
                }

                double final_distance;
                hipMemcpyAsync(&final_distance, &d_D.data[n * (m + 1) + m],
                                sizeof(double), hipMemcpyDeviceToHost);

                std::vector<int> h_path_i(h_path_length);
                std::vector<int> h_path_j(h_path_length);

                hipMemcpy(h_path_i.data(), d_path_i.data,
                           h_path_length * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(h_path_j.data(), d_path_j.data,
                           h_path_length * sizeof(int), hipMemcpyDeviceToHost);

                std::vector<std::pair<int, int>> path;
                path.reserve(h_path_length);
                for (int k = 0; k < h_path_length; ++k) {
                    path.emplace_back(h_path_i[k], h_path_j[k]);
                }

                return {final_distance, std::move(path)};
            }

            DTWResult dtw_cuda_impl(
                    const DoubleTimeSeries& A,
                    const DoubleTimeSeries& B,
                    distance::MetricType metric,
                    int tile_size) {

                switch(metric) {
                    case distance::MetricType::EUCLIDEAN:
                        return dtw_cuda_impl_template<distance::MetricType::EUCLIDEAN>(A, B, tile_size);
                    case distance::MetricType::MANHATTAN:
                        return dtw_cuda_impl_template<distance::MetricType::MANHATTAN>(A, B, tile_size);
                    case distance::MetricType::CHEBYSHEV:
                        return dtw_cuda_impl_template<distance::MetricType::CHEBYSHEV>(A, B, tile_size);
                    case distance::MetricType::COSINE:
                        return dtw_cuda_impl_template<distance::MetricType::COSINE>(A, B, tile_size);
                    default:
                        return dtw_cuda_impl_template<distance::MetricType::EUCLIDEAN>(A, B, tile_size);
                }
            }

            bool is_cuda_available() {
                int device_count = 0;
                hipError_t error = hipGetDeviceCount(&device_count);
                return error == hipSuccess && device_count > 0;
            }

            std::string get_cuda_device_info() {
                int device_count = 0;
                hipGetDeviceCount(&device_count);

                if (device_count == 0) {
                    return "No CUDA devices available";
                }

                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, 0);

                std::string info = "CUDA Device: ";
                info += prop.name;
                info += ", Compute Capability: ";
                info += std::to_string(prop.major) + "." + std::to_string(prop.minor);
                info += ", SMs: ";
                info += std::to_string(prop.multiProcessorCount);
                info += ", Max Threads/Block: ";
                info += std::to_string(prop.maxThreadsPerBlock);
                info += ", Shared Mem/Block: ";
                info += std::to_string(prop.sharedMemPerBlock) + " bytes";

                return info;
            }

        } // namespace cuda
    } // namespace parallel
} // namespace dtw_accelerator